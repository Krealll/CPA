#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <Windows.h>
#include <iostream>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <iomanip>

#pragma comment(lib, "hiprand.lib")

using namespace std;
#define SHOW_INITIAL_SIZE 16
#define SHOW_REORDERED_WIDTH  32
#define SHOW_REORDERED_HEIGHT  8
#define N 8000
#define M 2000
#define ELEMENTS_RANGE 5
#define BLOCK_DIM_X 32
#define BLOCK_DIM_Y 32
#define GRID_DIM_X  ((M + (BLOCK_DIM_X * 2 - 1))/(2*BLOCK_DIM_X) + 1)
#define GRID_DIM_Y (((N + (BLOCK_DIM_Y * 2 - 1))/(2*BLOCK_DIM_Y) + 1)

int hostTransitMatrix[N][M];
char hostInput[N][M];
char deviceInitial[N][M];
char hostCPUResult[N / 2][M * 2];
char deviceGPUResult[N / 2][M * 2];

void handleError(hipError_t error);
void handleStatus(hiprandStatus_t error);
void init(unsigned char* input, unsigned int* arr);
void mainCuda(unsigned char* devInput);
void reorder();
void checkEquality();
void showGPUResult(float time);

__global__ void kernel(unsigned char* input, unsigned char* result, unsigned int width, unsigned int height)
{
	int indexX = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
	int indexY = 2 * (blockIdx.y * blockDim.y + threadIdx.y);

	if (indexY < height && indexX < width)
	{
		char first = input[indexY *  width + indexX];
		char second = input[indexY *  width + indexX + 1];
		char third = input[(indexY + 1)*  width + indexX];
		char fourth = input[(indexY + 1)*  width + indexX + 1];

		result[indexY *  width + indexX * 2] = fourth;
		result[indexY *  width + indexX * 2 + 1] = second;
		result[indexY *  width + indexX * 2 + 2] = first;
		result[indexY *  width + indexX * 2 + 3] = third;
	}
}

int main()
{
	unsigned char* deviceInput;
	unsigned int* arr;
	handleError(hipMalloc((void**)& arr, N * M * sizeof(unsigned int)));
	handleError(hipMalloc((void**)& deviceInput, N * M * sizeof(unsigned char)));
	init(deviceInput, arr);
	mainCuda(deviceInput);


	reorder();
	checkEquality();
	handleError(hipFree(deviceInput));
	
	return 0;
}

void mainCuda(unsigned char* input)
{
	float GPUTime;
	hipEvent_t start, end;
	unsigned char * result;
	handleError(hipMalloc((void**)&result, N * M * sizeof(unsigned char)));
	dim3 _blockDim(BLOCK_DIM_X, BLOCK_DIM_Y);
	dim3 _gridDim(GRID_DIM_X, GRID_DIM_Y);


	handleError(hipEventCreate(&start));
	handleError(hipEventCreate(&end));
	handleError(hipEventRecord(start));

	kernel << <_gridDim, _blockDim >> > (input, result, M, N);

	handleError(hipEventRecord(end));
	handleError(hipEventSynchronize(end));
	handleError(hipEventElapsedTime(&GPUTime, start, end));


	handleError(hipMemcpy(deviceGPUResult, result, N * M * sizeof(char), hipMemcpyDeviceToHost));
	handleError(hipEventDestroy(start));
	handleError(hipEventDestroy(end));
	handleError(hipFree(result));
	showGPUResult(GPUTime);
}

void showGPUResult(float time)
{
	cout << "===================================================================================================" << endl;
	cout << "GPU work time: " << time << endl;
	cout << "===================================================================================================" << endl;
	cout << "GPU initial part:" << endl;
	for (int i = 0; i < SHOW_INITIAL_SIZE; i++)
	{
		for (int j = 0; j < SHOW_INITIAL_SIZE; j++)
		{
			cout << setw(ELEMENTS_RANGE) << deviceInitial[i][j];
		}
		cout << endl;
	}
	cout << "===================================================================================================" << endl;	cout << "GPU result part:" << endl;
	for (int i = 0; i < SHOW_REORDERED_HEIGHT; i++)
	{
		for (int j = 0; j < SHOW_REORDERED_WIDTH; j++)
		{
			cout << setw(ELEMENTS_RANGE) << deviceGPUResult[i][j];
		}
		cout << endl;
	}
	cout << "===================================================================================================" << endl;
}

void reorder()
{
	long long start, end;
	start = GetTickCount64();
	for (int i = 0, l = 0; i < N / 2; i++, l += 2)
	{
		for (int j = 0, k = 0; j < M * 2; j += 4, k += 2)
		{
			hostCPUResult[i][j + 0] = hostInput[l + 1][k + 1];
			hostCPUResult[i][j + 1] = hostInput[l][k + 1];
			hostCPUResult[i][j + 2] = hostInput[l][k];
			hostCPUResult[i][j + 3] = hostInput[l + 1][k];
		}
	}
	end = GetTickCount64();
	cout << "CPU reorder time: " << end - start << endl;
	cout << "===================================================================================================" << endl;
	cout << "CPU initial matrix part:" << endl;
	for (int i = 0; i < SHOW_INITIAL_SIZE; i++)
	{
		for (int j = 0; j < SHOW_INITIAL_SIZE; j++)
		{
			cout << setw(ELEMENTS_RANGE) << hostInput[i][j];
		}
		cout << endl;
	}
	cout << "===================================================================================================" << endl;
	cout << "CPU reordered matrix part:" << endl;
	for (int i = 0; i < SHOW_REORDERED_HEIGHT; i++)
	{
		for (int j = 0; j < SHOW_REORDERED_WIDTH; j++)
		{
			cout << setw(ELEMENTS_RANGE) << hostCPUResult[i][j];
		}
		cout << endl;
	}
}

void checkEquality()
{
	cout << "===================================================================================================" << endl;
	cout << "===================================================================================================" << endl;

	bool result = true;
	for (int i = 0; i < N / 2; i++)
	{
		for (int j = 0; j < M * 2; j++)
		{
			if (hostCPUResult[i][j] != deviceGPUResult[i][j])
			{
				result = false;
				//cout << i << " + " << j << " = " << hostCPUResult[i][j] << " != " << deviceGPUResult[i][j] <<endl;
			}
		}
	}
	if (result)
	{
		cout << "Results are equal" << endl;
	}
	else
	{
		cout << "NOT equal results!" << endl;
	}
	cout << "===================================================================================================" << endl;

}

void init(unsigned char* input, unsigned int* arr)
{
	hiprandGenerator_t generator;
	handleStatus(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
	handleStatus(hiprandSetPseudoRandomGeneratorSeed(generator, 1234ULL));
	handleStatus(hiprandGeneratePoisson(generator, arr, N * M, 50));
	handleError(hipMemcpy(hostTransitMatrix, arr, N * M * sizeof(unsigned int), hipMemcpyDeviceToHost));
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < M; j++)
		{
			hostInput[i][j] = *(char*)&hostTransitMatrix[i][j];
			deviceInitial[i][j] = *(char*)&hostTransitMatrix[i][j];
		}
	}
	handleError(hipMemcpy(input, hostInput, N * M * sizeof(unsigned char), hipMemcpyHostToDevice));
	handleStatus(hiprandDestroyGenerator(generator));
}

void handleError(hipError_t error)
{
	if (error != hipSuccess)
	{
		cout << "Error: " << hipGetErrorName(error) << endl;
		cout << "Error code - " << error << endl;
		cout << "Message - " << hipGetErrorString(error) << endl;
		exit(-1);
	}
}

void handleStatus(hiprandStatus_t status)
{
	if (status != HIPRAND_STATUS_SUCCESS)
	{
		cout << "hiprand status: " << status << endl;
	}
}
